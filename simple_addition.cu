#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<stdlib.h>

__global__ void add(int *d_a,int *d_b,int *d_sum)
{
    *d_sum = *d_a + *d_b;
}

int main()
{
    int a,b,sum;
    int *d_a,*d_b,*d_sum;
    int size = sizeof(int);
 
    a = 14;
    b = 23;
    sum = 0;
 
    hipMalloc((void**)&d_a,size);
    hipMalloc((void**)&d_b,size);
    hipMalloc((void**)&d_sum,size);
 
    hipMemcpy(d_a,&a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,&b,size,hipMemcpyHostToDevice);
    
    add<<<1,1>>>(d_a,d_b,d_sum);
 
    hipMemcpy(&sum,d_sum,size,hipMemcpyDeviceToHost);

    printf("Sum is : %d",sum);
 
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_sum);
    return 0;
}